#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


#define DEBUG 0



#define ENUM_NUM 19        // the number of loops in each thread
#define UNKNOWN_NUM 64     // the number of unknowns
#define POLY_NUM 64        // the number of linear polynomials
#define PARA_NUM 64	   // the number of parameters
#define NONLINEAR_NUM 64   // the number of nonlinear polynomials


#define SOL_MAX_NUM 200    
#define RESULT_MAX_NUM 5

// for GPU
#define BLOCK_NUM 32 //2^5
#define THREAD_NUM  256 // 2^8
#define THREADS_SHIFT 13 // (5+8)


typedef long value_t; // to save values of variables.
//typedef unsigned long constpart_t; // the part with no parameters.
typedef unsigned long linearpart_t; // to save 32 unknowns and 1 contants.
typedef unsigned long squarepart_t;
typedef unsigned long oripoly_t;
typedef unsigned char UINT8;
typedef unsigned long long UINT64;


//for GPU
__device__ linearpart_t d_linear_mat[ENUM_NUM * POLY_NUM * 2];
__device__ oripoly_t d_polys_mat[NONLINEAR_NUM * (POLY_NUM + UNKNOWN_NUM + 1) * 3];
__device__ squarepart_t d_square_mat[ENUM_NUM * POLY_NUM];
__device__ value_t d_var_all[2560];


/**
 * Print the number in binary, on CPU
 */
static inline void binary_print(value_t val, int len) {

    for (int i = 0; i < len; i++) {
        if (val & ((value_t)1 << i)) {
            printf("1");
        } else {
            printf("0");
        }

        if ((i + 1) % 5 == 0) {
            printf(" ");
        }
    }

}



/**
 * Find the position of the first nonzero bit, both on GPU and CPU 
 */
static inline __host__ __device__ int largestpos(value_t val, int len) {

	for (int i = len - 1; i >= 0; i--) {
		if (val & ((value_t) 1 << i)) {
			return i;
		}
	}

	return -1;
}


/**
 * Find the position of the first nonzero bit, both on GPU and CPU 
 *
 */
static inline __host__ __device__ int largestpos_2(value_t val0, value_t val1, int len) {
	int p = 0;
	if (len > 64 && len <= 128) {
		p = largestpos(val1, len - 64);
		if (p > -1) {
			return p + 64;
		} else {
			p = largestpos(val0, 64);

		}
	} else {
		p = largestpos(val0, 64);

	}

	return p;
}


/**
 * Solve the linear system by Guassian Elimination, on CPU
 */
static inline value_t gauss_host(linearpart_t working_mat[POLY_NUM][2],
		const int poly_num, const int unknown_num, value_t solutions[SOL_MAX_NUM]) {

	int pos_arr[POLY_NUM]; 
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
			continue;
		}


		pos_arr[pi] = largestpos_2(working_mat[pi][0],working_mat[pi][1], unknown_num + 1);

		rank++;
		if (pos_arr[pi] == 0) {
			return 0;
		}





		for (int j = pi + 1; j < POLY_NUM; j++) {

			if(working_mat[j][pos_arr[pi]/64] & ((linearpart_t)1 << (pos_arr[pi] % 64))){
                working_mat[j][0] ^= (working_mat[pi][0]);
                working_mat[j][1] ^= (working_mat[pi][1]);
            }
		}



	}


	// back reduced
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0] = 0;
;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 ) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t)1) {
				solutions[0] ^= ((value_t)1 << (pos_arr[pi]-1));
			}
		}

		return 1;

	} else {
		// multi-solutions
		solutions[0] = 0;
		value_t sol_num = 1;
		bool appear[UNKNOWN_NUM + 1] = { 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};



		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t)1) {
				solutions[0] ^= ((value_t)1 << (pos_arr[pi]-1));
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM+1; i++) {  

			if (appear[i] == false) {


				for (int j = 0; j < sol_num; j++) {
				    
                    solutions[j + sol_num] = (solutions[j]) ^ ((value_t)1 << (i-1));
				}

				
                for (int pi = 0; pi < POLY_NUM; pi++) {
				    if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 ) {
			            continue;
		            }

					if(i < 64){
						for (int j = 0; j < sol_num * ((working_mat[pi][0] & (((linearpart_t) 1) << i)) != 0); j++) {

			    		            solutions[j + sol_num] ^= ((value_t) 1 << (pos_arr[pi] - 1));
			    		   	}
					}else{
						for (int j = 0; j < sol_num * ((working_mat[pi][1] & (((linearpart_t) 1) << i-64)) != 0); j++) {

			    		            solutions[j + sol_num] ^= ((value_t) 1 << (pos_arr[pi] - 1));
			    		   	}
					}


				}


				sol_num *= 2;

			}
		}

		return sol_num;

	}

}



/**
 * Solve the linear system by Guassian Elimination, on GPU
 */
static inline __device__ value_t gauss(value_t solutions[SOL_MAX_NUM], linearpart_t working_mat[POLY_NUM][2],
		const int poly_num, const int unknown_num) {

	// bear revised
	int pos_arr[POLY_NUM]; // bear revised
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
			continue;
		}

		pos_arr[pi] = largestpos_2(working_mat[pi][0], working_mat[pi][1],unknown_num + 1);
		rank++;


		if (pos_arr[pi] == 0) {
			return 0;
		}

		for (int j = pi + 1; j < POLY_NUM; j++) {

			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
			}
		}

	}

	// back reduced
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0]= 0;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0] ^= ((value_t)1 << (pos_arr[pi]-1));
			}
		}

		return 1;

	} else {

	
		solutions[0] = 0;
		value_t sol_num = 1;
		
		bool appear[UNKNOWN_NUM + 1] = { 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0] ^= ((value_t)1 << (pos_arr[pi]-1));
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM + 1; i++) { 

			if (appear[i] == false) {

				for (int j = 0; j < sol_num; j++) {

				    solutions[j + sol_num] = (solutions[j]) ^ ((value_t)1 << (i-1));
				}


				for (int pi = 0; pi < POLY_NUM; pi++) {
					if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 ) {
						continue;
					}

					if(i < 64){
						for (int j = 0; j < sol_num * ((working_mat[pi][0] & (((linearpart_t) 1) << i)) != 0); j++) {

							solutions[j + sol_num] ^= ((value_t) 1 << (pos_arr[pi] - 1));
						}
					}else{
						for (int j = 0; j < sol_num * ((working_mat[pi][1] & (((linearpart_t) 1) << i-64)) != 0); j++) {

							 solutions[j + sol_num] ^= ((value_t) 1 << (pos_arr[pi] - 1));
						}
					}


				}
				sol_num *= 2;

			}
		}

		return sol_num;

	}

}




/**
 * Traverse the values of parameters to obtain matrices of linear system. Solve the system and count the numbers of satisfying equations.
 */
__global__ void solveLinear(const linearpart_t *d_working_mat_copy,
		const squarepart_t *d_const_mat, value_t *d_val, char *d_bound, value_t *d_sol_total,value_t* result) {

	int thidx = blockDim.x * blockIdx.x + threadIdx.x;
	value_t val = d_val[thidx];
	char bound = d_bound[thidx];
	value_t res[2 * RESULT_MAX_NUM];
	char res_num = 0;
	linearpart_t working_mat[POLY_NUM][2]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[POLY_NUM][2];

	squarepart_t const_mat[POLY_NUM];
	d_sol_total[thidx] = 0;
	oripoly_t cstpoly[3] = {0, 0, 0};

	//copy data from device
	for(int i = 0; i < 2 * RESULT_MAX_NUM; i++){
		res[i] = 0;
	}
	res[0] = result[thidx * 2 * RESULT_MAX_NUM];
	res[1] = result[thidx * 2 * RESULT_MAX_NUM + 1];


	for (int i = 0; i < POLY_NUM; i++) {
		working_mat_copy[i][0] = d_working_mat_copy[thidx * POLY_NUM * 2 + i*2];
		working_mat_copy[i][1] = d_working_mat_copy[thidx * POLY_NUM * 2 + i*2 + 1];

		const_mat[i] = d_const_mat[thidx * POLY_NUM + i];
	}



	// main loop.
	for (value_t count = 1; count < (1 << ENUM_NUM); count++) {

		// generate the next gray code
		int pos = 64-__ffsll(__brevll(count ^ (count - 1)));

		val = val ^ ((value_t) 1 << pos);


		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat_copy[pi][0] ^= d_linear_mat[pos * POLY_NUM * 2 + pi * 2];
			working_mat_copy[pi][1] ^= d_linear_mat[pos * POLY_NUM * 2 + pi * 2 + 1];
			const_mat[pi] ^= d_square_mat[pos * POLY_NUM + pi];

			working_mat[pi][0] = working_mat_copy[pi][0];
			working_mat[pi][1] = working_mat_copy[pi][1];


			value_t w = const_mat[pi] & val;


			working_mat[pi][0] ^= (bool)((__popcll((unsigned long long int)w)) & (value_t) 1);


		}




		value_t solutions[SOL_MAX_NUM];
		value_t sol_num = 0;


		// gaussian elimination
		sol_num = gauss(solutions, working_mat, POLY_NUM, UNKNOWN_NUM);
		d_sol_total[thidx] += sol_num;

		// verify on 3 round keccak
		for(int s = 0;s < sol_num;s++){
			int zero_num = 0;
			int one_num = 0;
			int op;

			for (op = 0; op < NONLINEAR_NUM; op++) {

				cstpoly[0] = d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + UNKNOWN_NUM) * 3];
				cstpoly[1] = d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + UNKNOWN_NUM) * 3 + 1];
				cstpoly[2] = d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + UNKNOWN_NUM) * 3 + 2];


				 	// for parameters.
					for (int pa = 0; pa < PARA_NUM; pa++) {
						if (val & ((value_t) 1 << pa)) {
							cstpoly[0] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM - pa -1) * 3];
							cstpoly[1] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM - pa -1) * 3 + 1];
							cstpoly[2] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM - pa -1) * 3 + 2];


						}
					}

					for (int un = 0; un < UNKNOWN_NUM; un++) {
						if (solutions[s] & ((value_t) 1 << un)) {
							cstpoly[0] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + un) * 3];
							cstpoly[1] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + un) * 3 + 1];
							cstpoly[2] ^= d_polys_mat[op * (PARA_NUM + UNKNOWN_NUM + 1) * 3 + (PARA_NUM + un) * 3 + 2];

						}
					}


					// evaluate
					cstpoly[0] = cstpoly[0] & val;
					cstpoly[1] = cstpoly[1] & solutions[s];
					cstpoly[2] = cstpoly[2] & ((oripoly_t)1);
					value_t w = cstpoly[0] ^ cstpoly[1] ^ cstpoly[2];

					w = (w) ^ (w >> 32);
					w = (w) ^ (w >> 16);
					w = (w) ^ (w >> 8);
					w = (w) ^ (w >> 4);
					w = (w) ^ (w >> 2);
					w = (w) ^ (w >> 1);


					if (w & (value_t) 1) {

					    zero_num ++;
					    if(zero_num > NONLINEAR_NUM - bound){
						break;
					    }
					}else{
					    one_num++;

					}


				}

			if(zero_num + one_num == NONLINEAR_NUM && one_num >= bound){

				if(one_num > bound){
					bound = one_num;
					res_num = 0;
					for(int ir = 0; ir < 2 * RESULT_MAX_NUM; ir++ ){
						res[ir] = 0;
					}
				}
				if(res_num < RESULT_MAX_NUM){
					res[res_num * 2] = val;
					res[res_num * 2 + 1 ] = solutions[s];
					res_num ++;
				}
//				    printf("\nval:%lu,sol:%lu,count:%d(bound:%d) thidx:%d  \n",val,solutions[s],one_num,bound, thidx);


			}

		}
	}

	d_bound[thidx] = bound;
	for(int i = 0; i < 2 * RESULT_MAX_NUM; i++){
		result[thidx * 2 * RESULT_MAX_NUM + i] = res[i];
	}

}






int main(int argc, char** argv) {


	char bound = 0;
	const int para_num = PARA_NUM;
	const int enum_num = ENUM_NUM;
	const int ori_num = NONLINEAR_NUM;


	value_t set_val = atol(argv[1])<<THREADS_SHIFT;


	const int poly_num = POLY_NUM;
	const int unknown_num = UNKNOWN_NUM;

	linearpart_t linear_mat[para_num][poly_num][2];
	linearpart_t working_mat[poly_num][2]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[poly_num][2];
	linearpart_t working_mat_file[poly_num][2];

	squarepart_t square_mat[para_num][poly_num];
	squarepart_t const_mat[poly_num]; // used to compute the const part from square polys.

        oripoly_t polys[ori_num][para_num + unknown_num + 1][3];
        oripoly_t cstpoly[3];

	
	hipSetDevice(atoi(argv[2]));  // set the GPU device 

	//read the matrix files
	FILE *in1 = fopen("./data/linear_mat224.txt", "r+");
	FILE *in2 = fopen("./data/square_mat224.txt", "r+");
	FILE *in3 = fopen("./data/polys_mat224.txt", "r+");
	FILE *in4 = fopen("./data/working_mat224.txt", "r+");


	char c1, c2, c3, c4;
	for (int i = 0; i < para_num; i++) {
		for (int j = 0; j < poly_num; j++) {
			linear_mat[i][j][0] = 0;
			linear_mat[i][j][1] = 0;
			square_mat[i][j] = 0;

			for (int k = 0; k < 128; k++) {
				fscanf(in1, "%c", &c1);
				while (c1 != '0' && c1 != '1') {
					fscanf(in1, "%c", &c1);
				}
				if (c1 == '1') {

					linear_mat[i][j][k/64] ^= ((linearpart_t) 1 << (k-((int)k/64)*64));
				}
			}
			for (int k = 0; k < para_num; k++) {
				fscanf(in2, "%c", &c2);
				while (c2 != '0' && c2 != '1') {
					fscanf(in2, "%c", &c2);
				}
				if (c2 == '1') {
					square_mat[i][j] ^= ((squarepart_t) 1 << (para_num - 1) - k);
				}
			}
		}
	}



	for (int i = 0; i < ori_num; i++) {
		for (int j = 0; j < para_num + unknown_num + 1; j++) {
			polys[i][j][0] = 0;
			polys[i][j][1] = 0;
			polys[i][j][2] = 0;
			for (int k = 0; k < 192; k++) {
				fscanf(in3, "%c", &c3);
				while (c3 != '0' && c3 != '1') {
					fscanf(in3, "%c", &c3);
				}
				if (k < para_num && c3 == '1') {
					polys[i][j][0] ^= ((oripoly_t) 1 << (para_num - k -1));
				} else if (k >= para_num && k<para_num+unknown_num && c3 == '1') {
					polys[i][j][1] ^= ((oripoly_t) 1 << (k - para_num));
				}else if(c3 == '1'){
					polys[i][j][2] ^= ((oripoly_t) 1);
				}
			}

		}

	}


	for (int i = 0; i < poly_num; i++) {
		working_mat[i][0] = 0;
		working_mat[i][1] = 0;
		for (int j = 0; j < 128; j++) {
			fscanf(in4, "%c", &c4);
			while (c4 != '0' && c4 != '1') {
				fscanf(in4, "%c", &c4);
			}
			if (c4 == '1') {

				working_mat[i][(int)j/64] ^= ((linearpart_t) 1 << (j - ((int)j/64)*64));
			}
		}

		working_mat_file[i][0] = working_mat[i][0];
		working_mat_file[i][1] = working_mat[i][1];


	}

	fclose(in1);
	fclose(in2);
	fclose(in3);
	fclose(in4);


	printf("finish reading file!\n");

	
	//allocate device memory
	linearpart_t linear_mat_enum[ENUM_NUM * POLY_NUM * 2];
	squarepart_t square_mat_enum[ENUM_NUM * POLY_NUM];
	oripoly_t polys_mat_enum[ori_num * (para_num + unknown_num + 1) * 3 ];

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			for (int k = 0; k < 2; k++) {
				linear_mat_enum[i * POLY_NUM * 2 + j * 2 + k] =
						linear_mat[i][j][k];

			}
		}
	}

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			square_mat_enum[i * POLY_NUM + j] = square_mat[i][j];
		}
	}

	for(int i = 0; i < ori_num; i++){
		for(int j = 0; j < para_num + unknown_num + 1; j++){
			for(int k = 0; k < 3; k++){
				polys_mat_enum[i * (para_num + unknown_num + 1) * 3 + j * 3 + k] = polys[i][j][k];
			}
		}
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_linear_mat), linear_mat_enum,
			2 * ENUM_NUM * POLY_NUM * sizeof(linearpart_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_square_mat), square_mat_enum,
			ENUM_NUM * POLY_NUM * sizeof(squarepart_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_polys_mat), polys_mat_enum,
				3 * NONLINEAR_NUM * (PARA_NUM + UNKNOWN_NUM + 1)  * sizeof(oripoly_t));

	printf("finish copying device memory!\n");

	hipError_t err = hipSuccess;
	int thidx = BLOCK_NUM * THREAD_NUM;

	value_t *d_val = NULL;
	err = hipMalloc((void **) &d_val, thidx * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	char *d_bound = NULL;
		err = hipMalloc((void **) &d_bound, thidx * sizeof(char));
		if (err != hipSuccess) {
			printf("Failed to allocate device value (error code %s)!\n",
					hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

	value_t *d_sol_total = NULL;
	err = hipMalloc((void **) &d_sol_total, thidx * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	linearpart_t *d_working_mat_copy = NULL;
	err = hipMalloc((void **) &d_working_mat_copy,
			thidx * poly_num * 2 * sizeof(linearpart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate device working_mat_copy (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	squarepart_t *d_const_mat = NULL;
	err = hipMalloc((void **) &d_const_mat,
			thidx * poly_num * sizeof(squarepart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices const_mat (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy oripolys from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t h_result[thidx * RESULT_MAX_NUM * 2];
	for(int i = 0; i < thidx * RESULT_MAX_NUM * 2 ; i ++){

		h_result[i] = 0;
	}




	printf("finish allocate device memory!\n");

	//deal with the case with all parameters are 0's on CPU
	int s_total_p0 = 0;
	value_t *val_arr = (value_t*)calloc(thidx, sizeof(value_t));
	linearpart_t *working_mat_copy_arr = (linearpart_t*)calloc(thidx * POLY_NUM * 2, sizeof(linearpart_t));
	squarepart_t *const_mat_arr = (squarepart_t*)calloc(thidx * POLY_NUM, sizeof(squarepart_t));
	char *bound_arr = (char*)calloc(thidx, sizeof(short));

	for (int thi = 0; thi < thidx; thi++) {

		value_t sol_num = 0;
		value_t solutions[SOL_MAX_NUM];
		value_t val = (set_val + (value_t) thi) << ENUM_NUM;
		val_arr[thi] = val;

		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat[pi][0] = working_mat_file[pi][0];
			working_mat[pi][1] = working_mat_file[pi][1];
			const_mat[pi] = 0;
		}

		for (int pos = enum_num; pos < para_num; pos++) {

			if (val & ((value_t) 1 << pos)) {

				for (int pi = 0; pi < poly_num; pi++) {
					working_mat[pi][0] ^= linear_mat[pos][pi][0];
					working_mat[pi][1] ^= linear_mat[pos][pi][1];
				}

				for (int pi = 0; pi < poly_num; pi++) {
					const_mat[pi] ^= square_mat[pos][pi];

				}

			}

		}


		for (int i = 0; i < poly_num; i++) {
			working_mat_copy[i][0] = working_mat[i][0];
			working_mat_copy[i][1] = working_mat[i][1];

			working_mat_copy_arr[thi * POLY_NUM * 2 + 2 * i] = working_mat_copy[i][0];
			working_mat_copy_arr[thi * POLY_NUM * 2 + 2 * i + 1] = working_mat_copy[i][1];

			const_mat_arr[thi * POLY_NUM + i] = const_mat[i];

		}

		for (int pi = 0; pi < poly_num; pi++) {

			value_t w = const_mat[pi] & val;

			w = (w) ^ (w >> 32);
			w = (w) ^ (w >> 16);
			w = (w) ^ (w >> 8);
			w = (w) ^ (w >> 4);
			w = (w) ^ (w >> 2);
			w = (w) ^ (w >> 1);

			if (w & (value_t) 1) {

				working_mat[pi][0] ^= (linearpart_t) 1;
			}

		}

		sol_num = gauss_host(working_mat, POLY_NUM, UNKNOWN_NUM, solutions);
		s_total_p0 += sol_num;

		//verify the solutions
		for (int s = 0; s < sol_num; s++) {
			int one_num = 0;
			int zero_num = 0;
			int op;
			for (op = 0; op < ori_num; op++) {

				cstpoly[0] = polys[op][para_num + unknown_num][0];
				cstpoly[1] = polys[op][para_num + unknown_num][1];
				cstpoly[2] = polys[op][para_num + unknown_num][2];
				// for parameters.
				for (int pa = 0; pa < para_num; pa++) {
					if (val & ((value_t) 1 << pa)) {

						cstpoly[0] ^= polys[op][para_num - pa - 1][0];
						cstpoly[1] ^= polys[op][para_num - pa - 1][1];
						cstpoly[2] ^= polys[op][para_num - pa - 1][2];
					}
				}

				for (int un = 0; un < unknown_num; un++) {
					if (solutions[s] & ((value_t) 1 << un)) {
						cstpoly[0] ^= polys[op][para_num + un][0];
						cstpoly[1] ^= polys[op][para_num + un][1];
						cstpoly[2] ^= polys[op][para_num + un][2];
					}
				}

				cstpoly[2] ^= polys[op][unknown_num + para_num][2];
				// evaluate
				cstpoly[0] = cstpoly[0] & val;
				cstpoly[1] = cstpoly[1] & solutions[s];
				cstpoly[2] = cstpoly[2] & ((oripoly_t) 1);

				value_t w = cstpoly[0] ^ cstpoly[1] ^ cstpoly[2];

				w = (w) ^ (w >> 32);
				w = (w) ^ (w >> 16);
				w = (w) ^ (w >> 8);
				w = (w) ^ (w >> 4);
				w = (w) ^ (w >> 2);
				w = (w) ^ (w >> 1);

				if (w & (value_t) 1) {
					zero_num++;
					if (zero_num > ori_num - bound) {
						break;
					}

				} else {
					one_num++;
				}

			}
			if (zero_num + one_num == ori_num) {
				printf( "val:%lu,sol:%lu,count:%d\n", val, solutions[s],one_num);
				h_result[thi * RESULT_MAX_NUM * 2] = val;
				h_result[thi * RESULT_MAX_NUM * 2 + 1] = solutions[s];

			}
			if (one_num > bound) {
				bound = one_num;
				bound_arr[thi] = bound;
			}

		}

	}

	for(int i = 0; i < thidx;i++){
		if(bound_arr[i] < bound){
			bound_arr[i] = bound;
			h_result[i * RESULT_MAX_NUM * 2] = 0;
			h_result[i * RESULT_MAX_NUM * 2 + 1] = 0;

		}
	}



	printf("finish cpu computing! the bound is %d now...\n", bound);

	//begin device part: copy value from host ot devide
	err = hipMemcpy(d_val, val_arr, thidx * sizeof(value_t),
			hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Failed to copy value from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_bound, bound_arr, thidx * sizeof(char),
			hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Failed to copy bound from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t *d_result = NULL;
	err = hipMalloc((void **) &d_result,  thidx * RESULT_MAX_NUM * 2 * sizeof(value_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices result (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_result, h_result,
				thidx * RESULT_MAX_NUM * 2 * sizeof(value_t),hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy result from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_working_mat_copy, working_mat_copy_arr,
			thidx * 2 * poly_num * sizeof(linearpart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy working_mat_copy from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_const_mat, const_mat_arr,
			thidx * poly_num * sizeof(squarepart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy const_mat from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("enum num : %d\nblock num : %d\nthread num : %d\n", ENUM_NUM,
			BLOCK_NUM, THREAD_NUM);

	//solve linear system on GPU
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	printf("begin solve linear system!\n");
	solveLinear<<<BLOCK_NUM, THREAD_NUM>>>(d_working_mat_copy, d_const_mat,
			d_val, d_bound, d_sol_total,d_result);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch solveLinear kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	value_t h_sol_total[thidx];
	err = hipMemcpy(h_sol_total, d_sol_total, thidx * sizeof(value_t),
				hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy total solution numbers from device to host (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
	err = hipMemcpy(h_result, d_result, thidx * RESULT_MAX_NUM * 2 * sizeof(value_t),
						hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy result from device to host (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(bound_arr, d_bound, thidx * sizeof(char),
					hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
					"Failed to copy bound from device to host (error code %s)!\n",
					hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	//sort the results

	for(int i = 0; i < thidx; i++){
		for(int j = i + 1; j < thidx; j++){
			if(bound_arr[i] > bound_arr[j]){
				char temp = bound_arr[i];
				bound_arr[i] = bound_arr[j];
				bound_arr[j] = temp;

				for(int ri = 0;ri < 2 * RESULT_MAX_NUM; ri ++){
					value_t temp = h_result[i * 2 * RESULT_MAX_NUM + ri];
					h_result[i * 2 * RESULT_MAX_NUM + ri] = h_result[j * 2 * RESULT_MAX_NUM + ri];
					h_result[j * 2 * RESULT_MAX_NUM + ri] = temp;
				}

			}
		}
	}

//	printf("\n------------ finish sort ---------------\n");

	//write the result of each threads 
	printf("bound:%d\n",bound_arr[thidx-1]);

	FILE *out = fopen("result.txt","a+");

	for(int i = thidx - 1;i >= 0;i--){
		fprintf(out,"bound:%d\n",bound_arr[i]);
		for(int j = 0; j < RESULT_MAX_NUM; j++){
			fprintf(out,"%d. val:%lu, sol:%lu\n",j+1, h_result[i * 2 * RESULT_MAX_NUM + j * 2],h_result[i*2 * RESULT_MAX_NUM + j * 2 + 1]);
		}
		if(bound_arr[i] > bound_arr[i-1]){
			break;
		}
	}


	printf("time:%.3lf ms\n---------------------------------------\n", msecTotal1);


	hipFree(d_working_mat_copy);
	hipFree(d_const_mat);
	hipFree(d_val);
	hipFree(d_bound);
	hipFree(d_sol_total);
	hipFree(d_result);
}
