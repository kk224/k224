#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

//#include <helper_cuda.h>

#define DEBUG 0


#define ENUM_NUM 16        // the number of loops in each thread
#define UNKNOWN_NUM 187    // the number of unknowns
#define POLY_NUM 190       // the number of linear polynomials
#define PARA_NUM 37        // the number of parameters
#define CHECK_NUM 9       
#define SOL_MAX_NUM 200


//for GPU
#define BLOCK_NUM 32 //2^5
#define THREAD_NUM  256 // 2^8
#define THREADS_SHIFT 13 // (5+8)


//for keccak
#define maxNrRounds 24
#define nrLanes 25
#define index(x, y) (((x)%5)+5*((y)%5))
#define KeccakP1600_stateSizeInBytes    200

typedef long value_t; // to save values of variables.
typedef unsigned long linearpart_t; // to save 32 unknowns and 1 contants.
typedef unsigned long squarepart_t;
typedef unsigned long oripoly_t;
typedef unsigned char UINT8;
typedef unsigned long long UINT64;
typedef UINT64 tKeccakLane;


static inline void binary_print(value_t val, int len) {

    for (int i = 0; i < len; i++) {
        if (val & ((value_t)1 << i)) {
            printf("1");
        } else {
            printf("0");
        }

        if ((i + 1) % 5 == 0) {
            printf(" ");
        }
    }

}

//Operations of keccak. On CPU

static tKeccakLane KeccakRoundConstants[maxNrRounds];
static unsigned int KeccakRhoOffsets[nrLanes];

__constant__ tKeccakLane const_KeccakRoundConstants[maxNrRounds] =
{
    0x0000000000000001,
    0x0000000000008082,
    0x800000000000808a,
    0x8000000080008000,
    0x000000000000808b,
    0x0000000080000001,
    0x8000000080008081,
    0x8000000000008009,
    0x000000000000008a,
    0x0000000000000088,
    0x0000000080008009,
    0x000000008000000a,
    0x000000008000808b,
    0x800000000000008b,
    0x8000000000008089,
    0x8000000000008003,
    0x8000000000008002,
    0x8000000000000080,
    0x000000000000800a,
    0x800000008000000a,
    0x8000000080008081,
    0x8000000000008080,
    0x0000000080000001,
    0x8000000080008008,
};


__constant__ unsigned int const_KeccakRhoOffsets[nrLanes] =
{
     0,  1, 62, 28, 27, 36, 44,  6, 55, 20,  3, 10, 43, 25, 39, 41, 45, 15, 21,  8, 18,  2, 61, 56, 14
};


void KeccakP1600_InitializeRoundConstants(void);
void KeccakP1600_InitializeRhoOffsets(void);
static int LFSR86540(UINT8 *LFSR);
int displayLevel = 10;


 void KeccakP1600_StaticInitialize(void)
{
    if (sizeof(tKeccakLane) != 8) {
        printf("tKeccakLane should be 64-bit wide\n");
        abort();
    }
    KeccakP1600_InitializeRoundConstants();
    KeccakP1600_InitializeRhoOffsets();
}


 void KeccakP1600_InitializeRoundConstants(void)
{
    UINT8 LFSRstate = 0x01;
    unsigned int i, j, bitPosition;

    for(i=0; i<maxNrRounds; i++) {
        KeccakRoundConstants[i] = 0;
        for(j=0; j<7; j++) {
            bitPosition = (1<<j)-1; /* 2^j-1 */
            if (LFSR86540(&LFSRstate))
                KeccakRoundConstants[i] ^= (tKeccakLane)1<<bitPosition;
        }
    }
}


 void KeccakP1600_InitializeRhoOffsets(void)
{
    unsigned int x, y, t, newX, newY;

    KeccakRhoOffsets[index(0, 0)] = 0;
    x = 1;
    y = 0;
    for(t=0; t<24; t++) {
        KeccakRhoOffsets[index(x, y)] = ((t+1)*(t+2)/2) % 64;
        newX = (0*x+1*y) % 5;
        newY = (2*x+3*y) % 5;
        x = newX;
        y = newY;
    }
}


static int LFSR86540(UINT8 *LFSR)
{
    int result = ((*LFSR) & 0x01) != 0;
    if (((*LFSR) & 0x80) != 0)
    /* Primitive polynomial over GF(2): x^8+x^6+x^5+x^4+1 */
        (*LFSR) = ((*LFSR) << 1) ^ 0x71;
    else
        (*LFSR) <<= 1;
    return result;
}

//Operations of keccak. on CPU and GPU
__host__ __device__ void KeccakP1600_Initialize(void *state)
{
    memset(state, 0, 1600/8);
}


/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_AddByte(void *state, unsigned char byte, unsigned int offset)
{
    assert(offset < 200);
    ((unsigned char *)state)[offset] ^= byte;
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_AddBytes(void *state, const unsigned char *data, unsigned int offset, unsigned int length)
{
    unsigned int i;

    assert(offset < 200);
    assert(offset+length <= 200);
    for(i=0; i<length; i++)
        ((unsigned char *)state)[offset+i] ^= data[i];
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_OverwriteBytes(void *state, const unsigned char *data, unsigned int offset, unsigned int length)
{
    assert(offset < 200);
    assert(offset+length <= 200);
    memcpy((unsigned char*)state+offset, data, length);
}

/* ---------------------------------------------------------------- */

__host__ __device__ void KeccakP1600_OverwriteWithZeroes(void *state, unsigned int byteCount)
{
    assert(byteCount <= 200);
    memset(state, 0, byteCount);
}


#define ROL64(a, offset) ((offset != 0) ? ((((tKeccakLane)a) << offset) ^ (((tKeccakLane)a) >> (64-offset))) : a)

static __host__ __device__ void theta(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane C[5]={0,0,0,0,0}, D[5]={0,0,0,0,0};

    for(x=0; x<5; x++) {
        C[x] = 0;
        for(y=0; y<5; y++)
            C[x] ^= A[index(x, y)];
    }
    for(x=0; x<5; x++)
        D[x] = ROL64(C[(x+1)%5], 1) ^ C[(x+4)%5];
    for(x=0; x<5; x++)
        for(y=0; y<5; y++)
            A[index(x, y)] ^= D[x];
}


static void rho(tKeccakLane *A)
{
    unsigned int x, y;

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(x, y)] = ROL64(A[index(x, y)], KeccakRhoOffsets[index(x, y)]);
}

static __device__ void rho_Device(tKeccakLane *A)
{
    unsigned int x, y;

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(x, y)] = ROL64(A[index(x, y)], const_KeccakRhoOffsets[index(x, y)]);
}

static __host__ __device__ void pi(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane tempA[25];

    for(x=0; x<5; x++) for(y=0; y<5; y++)
        tempA[index(x, y)] = A[index(x, y)];
    for(x=0; x<5; x++) for(y=0; y<5; y++)
        A[index(0*x+1*y, 2*x+3*y)] = tempA[index(x, y)];
}

static __host__ __device__ void chi(tKeccakLane *A)
{
    unsigned int x, y;
    tKeccakLane C[5];

    for(y=0; y<5; y++) {
        for(x=0; x<5; x++)
            C[x] = A[index(x, y)] ^ ((~A[index(x+1, y)]) & A[index(x+2, y)]);
        for(x=0; x<5; x++)
            A[index(x, y)] = C[x];
    }
}

static void iota(tKeccakLane *A, unsigned int indexRound)
{
    A[index(0, 0)] ^= KeccakRoundConstants[indexRound];
}

static __device__ void iota_Device(tKeccakLane *A, unsigned int indexRound)
{
    A[index(0, 0)] ^= const_KeccakRoundConstants[indexRound];
}

void KeccakP1600Round(tKeccakLane *state, unsigned int indexRound)
{
#ifdef KeccakReference
    printf("11\n");
    displayRoundNumber(3, indexRound);
#endif

    theta(state);

#ifdef KeccakReference
    displayStateAsLanes(3, "After theta", state, 1600);
#endif

    rho(state);
#ifdef KeccakReference
    displayStateAsLanes(3, "After rho", state, 1600);
#endif

    pi(state);
#ifdef KeccakReference
    displayStateAsLanes(3, "After pi", state, 1600);
#endif

    chi(state);
#ifdef KeccakReference
    displayStateAsLanes(3, "After chi", state, 1600);
#endif

    iota(state, indexRound);
#ifdef KeccakReference
    displayStateAsLanes(3, "After iota", state, 1600);
#endif
}



__device__ void KeccakP1600Round_Device(tKeccakLane *state,unsigned int indexRound) {

	theta(state);
	rho_Device(state);
	pi(state);
	chi(state);
	iota_Device(state, indexRound);

}


/**
 * Initial the state from the output of block 1
 */
void stateInit(tKeccakLane state[25]) {

	KeccakP1600_StaticInitialize();
	for(int i = 0; i < 5; i++){
		state[i] = 0;
		state[i+10]=0;
	}
        state[18] = 0x90568729AB3FF556;
	state[19] = 0x16D513824AA92784;
	state[20] = 0x04A0F3940480011D;
	state[21] = 0x0A87CA1C3B68EF23;
	state[22] = 0xF5A858BD84F680F6;
	state[23] = 0x63A9789654C00AA9;
	state[24] = 0xE128EC7DB556D87B;
	for (int i = 0; i < 5; i++) {
		state[5 + i] = ~(state[20 + i]);
		if (i < 3) {
			state[15 + i] = ~(state[20 + i]);
		}
	}


}


/*
 * Obtain the initial state from the solutions 
 */

void getStates(tKeccakLane state[25], oripoly_t var_all[640][4], value_t val,
		value_t solutions[3]) {

	value_t val_sol[4];
	val_sol[3] = solutions[2];
	val_sol[2] = solutions[1];
	val_sol[1] = solutions[0];
	val_sol[0] = val ^ ((value_t) 1 << 37);

	for (int i = 0; i < 640; i++) {
		value_t w[4] = { 0, 0, 0, 0 };
		for (int j = 0; j < 4; j++) {
			w[j] = var_all[i][j] & val_sol[j];

		}

		w[0] = w[0] ^ w[1] ^ w[2] ^ w[3];
		w[0] = (w[0]) ^ (w[0] >> 32);
		w[0] = (w[0]) ^ (w[0] >> 16);
		w[0] = (w[0]) ^ (w[0] >> 8);
		w[0] = (w[0]) ^ (w[0] >> 4);
		w[0] = (w[0]) ^ (w[0] >> 2);
		w[0] = (w[0]) ^ (w[0] >> 1);
		if (w[0] & (value_t) 1) {
			int n = (i / 64 > 4) ? (i / 64 + 5) : i / 64;
			state[n] ^= ((UINT64) 1) << (i % 64);

		}
	}


}

/**
 *  Verify solutions
 */

int checkHashValue(tKeccakLane state[25]) {


	tKeccakLane state_copy[25];
	for(int i = 0; i < 25; i++){
		state_copy[i] = state[i];
	}





	for (int i = 0; i < 3; i++) {
		KeccakP1600Round(state, i);
	}



	int result = 0;

    	if (state[0] == 0xF4FE7CCEA5D8B144 && state[1] == 0x60F6C316572983A8 && state[2] == 0xA2564CA289E5F897 && ((state[3] ^= 0x00000000CA30DB85) & (0x00000000FFFFFFFF)) == 0) {

		printf("Find Preimage!!!\nState after XORed with block2:");
		for (int i = 0; i < 25; i++) {
			binary_print(state_copy[i], 64);
			printf(" ");
			printf("%llu ",state_copy[i]);
		}
		printf("\n");


		result = 1;
	}
	return result;
}


//the output of block 1 used on GPU
__constant__ tKeccakLane const_state[25] = {0x0L,0x0L,0x0L,0x0L,0x0L,
        0xFB5F0C6BFB7FFEE2L,0xF57835E3C49710DCL,0x0A57A7427B097F09L,0x9C568769AB3FF556L,0x1ED713824AA92784L,
        0x0L,0x0L,0x0L,0x0L,0x0L,
        0xFB5F0C6BFB7FFEE2L,0xF57835E3C49710DCL,0x0A57A7427B097F09L,0x90568729AB3FF556L,0x16D513824AA92784L,
        0x04A0F3940480011DL,0x0A87CA1C3B68EF23L,0xF5A858BD84F680F6L,0x63A9789654C00AA9L,0xE128EC7DB556D87BL};


__device__ linearpart_t d_linear_mat[ENUM_NUM * POLY_NUM * 3];
__device__ squarepart_t d_square_mat[ENUM_NUM * POLY_NUM];
__device__ value_t d_var_all[2560];


/**
 * Find the position of the first nonzero bit, both on GPU and CPU 
 */
static inline __host__ __device__ int largestpos(value_t val, int len) {

	for (int i = len - 1; i >= 0; i--) {
		if (val & ((value_t) 1 << i)) {
			return i;
		}
	}

	return -1;
}

/**
 * Find the position of the first nonzero bit, both on GPU and CPU 
 */
static inline __host__ __device__ int largestpos_3(value_t val0, value_t val1,
		value_t val2, int len) {
	int p = 0;
	if (len > 128) {
		p = largestpos(val2, len - 128);
		if (p > -1) {
			return p + 128;
		} else {
			p = largestpos(val1, 64);
			if (p > -1) {
				return p + 64;
			} else {
				p = largestpos(val0, 64);

			}
		}
	} else if (len > 64 && len <= 128) {
		p = largestpos(val1, len - 64);
		if (p > -1) {
			return p + 64;
		} else {
			p = largestpos(val0, 64);

		}
	} else {
		p = largestpos(val0, 64);

	}

	return p;
}

/**
 * Solve the linear system by Guassian Elimination, on CPU
 */
static inline value_t gauss_host(linearpart_t working_mat[POLY_NUM][3],
		const int poly_num, const int unknown_num, value_t solutions[SOL_MAX_NUM][3]) {

	int pos_arr[POLY_NUM]; 
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
			continue;
		}


		pos_arr[pi] = largestpos_3(working_mat[pi][0],working_mat[pi][1],working_mat[pi][2], unknown_num + 1);

		rank++;
		if (pos_arr[pi] == 0) {
			return 0;
		}





		for (int j = pi + 1; j < POLY_NUM; j++) {

			if(working_mat[j][pos_arr[pi]/64] & ((linearpart_t)1 << (pos_arr[pi] % 64))){
                working_mat[j][0] ^= (working_mat[pi][0]);
                working_mat[j][1] ^= (working_mat[pi][1]);
                working_mat[j][2] ^= (working_mat[pi][2]);
            }
		}



	}


	// back reduced
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 &&  working_mat[pi][2] == 0) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t)1) {
			    solutions[0][(pos_arr[pi]-1) /64 ] ^= ((value_t)1 << (pos_arr[pi]-1) % 64);
			}
		}

		return 1;

	} else {

		// multi-solutions
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		value_t sol_num = 1;
		bool appear[UNKNOWN_NUM + 1] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};



		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t)1) {
			    solutions[0][(pos_arr[pi]-1) /64 ] ^= ((value_t)1 << (pos_arr[pi]-1) % 64);
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM+1; i++) {  

			if (appear[i] == false) {


				for (int j = 0; j < sol_num; j++) {
				    
					solutions[j + sol_num][0] = solutions[j][0];
					solutions[j + sol_num][1] = solutions[j][1];
					solutions[j + sol_num][2] = solutions[j][2];
					solutions[j + sol_num][(i-1)/64] ^= ((value_t)1 << ((i-1)%64));
				}


                for (int pi = 0; pi < POLY_NUM; pi++) {
				    if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0 && working_mat[pi][2] == 0) {
			            continue;
		            }

				    for (int j = 0; j < sol_num * ((working_mat[pi][i/64] & (((linearpart_t) 1) << (i%64))) != 0); j++) {

				    	solutions[j + sol_num][(pos_arr[pi] - 1)/64] ^= ((value_t) 1 << ((pos_arr[pi] - 1)% 64));
				    }


				}


				sol_num *= 2;

			}
		}

		return sol_num;

	}

}



/**
 * Solve the linear system by Guassian Elimination, on GPU
 */
static inline __device__ value_t gauss(value_t solutions[SOL_MAX_NUM][3], linearpart_t working_mat[POLY_NUM][3],
		const int poly_num, const int unknown_num) {

	// bear revised
	int pos_arr[POLY_NUM]; // bear revised
	int rank = 0;

	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		pos_arr[pi] = largestpos_3(working_mat[pi][0], working_mat[pi][1],working_mat[pi][2],unknown_num + 1);
		rank++;


		if (pos_arr[pi] == 0) {
			return 0;
		}

		for (int j = pi + 1; j < POLY_NUM; j++) {

			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}

	}

	// back
	for (int pi = 0; pi < POLY_NUM; pi++) {

		if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
				&& working_mat[pi][2] == 0) {
			continue;
		}

		for (int j = 0; j < pi; j++) {
			if (working_mat[j][pos_arr[pi] / 64]
					& ((linearpart_t) 1 << (pos_arr[pi] % 64))) {
				working_mat[j][0] ^= (working_mat[pi][0]);
				working_mat[j][1] ^= (working_mat[pi][1]);
				working_mat[j][2] ^= (working_mat[pi][2]);
			}
		}
	}

	if (rank == unknown_num) {

		// only one solution.
		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
					&& working_mat[pi][2] == 0) {
				continue;
			}
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0][(pos_arr[pi] - 1) / 64] ^= ((value_t) 1
						<< (pos_arr[pi] - 1) % 64);
			}
		}

		return 1;

	} else {


		solutions[0][0] = 0;
		solutions[0][1] = 0;
		solutions[0][2] = 0;
		value_t sol_num = 1;
	
		bool appear[UNKNOWN_NUM + 1] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
				0, 0, 0, 0, 0 };

		for (int pi = 0; pi < POLY_NUM; pi++) {

			if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
					&& working_mat[pi][2] == 0) {
				continue;
			}

			appear[pos_arr[pi]] = true;
			if (working_mat[pi][0] & (linearpart_t) 1) {
				solutions[0][(pos_arr[pi] - 1) / 64] ^= ((value_t) 1
						<< (pos_arr[pi] - 1) % 64);
			}
		}

		// duplicate solutions.
		for (int i = 1; i < UNKNOWN_NUM + 1; i++) {  
			if (appear[i] == false) {

				for (int j = 0; j < sol_num; j++) {

					solutions[j + sol_num][0] = solutions[j][0];
					solutions[j + sol_num][1] = solutions[j][1];
					solutions[j + sol_num][2] = solutions[j][2];
					solutions[j + sol_num][(i - 1) / 64] ^= ((value_t) 1
							<< ((i - 1) % 64));
				}


				for (int pi = 0; pi < POLY_NUM; pi++) {
					if (working_mat[pi][0] == 0 && working_mat[pi][1] == 0
							&& working_mat[pi][2] == 0) {
						continue;
					}

					for (int j = 0;j< sol_num* ((working_mat[pi][i / 64]& (((linearpart_t) 1)<< (i % 64))) != 0);j++) {

						solutions[j + sol_num][(pos_arr[pi] - 1) / 64] ^=((value_t) 1 << ((pos_arr[pi] - 1) % 64));
					}

				}

				sol_num *= 2;

			}
		}

		return sol_num;

	}

}




/**
 * Traverse the values of parameters to obtain matrices of linear system. Solve the system and verify the solutions.
 */
__global__ void solveLinear(const linearpart_t *d_working_mat_copy,
		const squarepart_t *d_const_mat, value_t *d_val, value_t *d_sol_total,value_t* result) {

	int thidx = blockDim.x * blockIdx.x + threadIdx.x;
	value_t val = d_val[thidx];

	linearpart_t working_mat[POLY_NUM][3]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[POLY_NUM][3];

	squarepart_t const_mat[POLY_NUM];
	d_sol_total[thidx] = 0;



	//copy data from device
	for (int i = 0; i < POLY_NUM; i++) {
		working_mat_copy[i][0] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3];
		working_mat_copy[i][1] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3 + 1];
		working_mat_copy[i][2] = d_working_mat_copy[thidx * POLY_NUM * 3 + i*3 + 2];

		const_mat[i] = d_const_mat[thidx * POLY_NUM + i];
	}

	// main loop.
	for (value_t count = 1; count < (1 << ENUM_NUM); count++) {

		// generate the next gray code
		int pos = 64-__ffsll(__brevll(count ^ (count - 1)));

		val = val ^ ((value_t) 1 << pos);


		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat_copy[pi][0] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3];
			working_mat_copy[pi][1] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3 + 1];
			working_mat_copy[pi][2] ^= d_linear_mat[pos * POLY_NUM * 3 + pi * 3 + 2];

			const_mat[pi] ^= d_square_mat[pos * POLY_NUM + pi];

			working_mat[pi][0] = working_mat_copy[pi][0];
			working_mat[pi][1] = working_mat_copy[pi][1];
			working_mat[pi][2] = working_mat_copy[pi][2];


			value_t w = const_mat[pi] & val;


			working_mat[pi][0] ^= (bool)((__popcll((unsigned long long int)w)) & (value_t) 1);


		}

		value_t solutions[SOL_MAX_NUM][3];
		value_t sol_num = 0;


		// gaussian elimination
		sol_num = gauss(solutions, working_mat, POLY_NUM, UNKNOWN_NUM);
		d_sol_total[thidx] += sol_num;

		// verify on 3 round keccak.
		tKeccakLane dState[25];


		for(int s = 0;s < sol_num;s++){
			dState[0] = 0;
			dState[1] = 0;
			dState[2] = 0;
			dState[3] = 0;
			dState[4] = 0;
			dState[5] = const_state[5];
			dState[6] = const_state[6];
			dState[7] = const_state[7];
			dState[8] = const_state[8];
			dState[9] = const_state[9];
			dState[10] = 0;
			dState[11] = 0;
			dState[12] = 0;
			dState[13] = 0;
			dState[14] = 0;
			dState[15] = const_state[15];
			dState[16] = const_state[16];
			dState[17] = const_state[17];
			dState[18] = const_state[18];
			dState[19] = const_state[19];

			dState[20] = const_state[20];
			dState[21] = const_state[21];
			dState[22] = const_state[22];
			dState[23] = const_state[23];
			dState[24] = const_state[24];


			value_t val_sol[4];
			val_sol[3] = solutions[s][2];
			val_sol[2] = solutions[s][1];
			val_sol[1] = solutions[s][0];
			val_sol[0] = val ^ ((value_t)1 << 37);

			for(int i = 0; i< 640; i ++){
				value_t w[4] ={0,0,0,0};
				for(int j = 0; j< 4; j++){
					w[j] = d_var_all[i * 4 + j] & val_sol[j];
				}

				w[0] = w[0] ^w[1]^w[2]^w[3];


				if ((bool)((__popcll((unsigned long long int)w[0])) & (value_t) 1)) {
					int n = (i/64 > 4 )?( i/64 + 5 ): i/64 ;
					dState[n] ^= ((UINT64)1) << (i % 64);
				}
			}


			tKeccakLane state_copy[25];
			for(int i = 0; i < 25; i++){
				state_copy[i] = dState[i];
			}



            KeccakP1600Round_Device(dState, 0);

            KeccakP1600Round_Device(dState, 1);
            KeccakP1600Round_Device(dState, 2);



            if (dState[0] == 0xF4FE7CCEA5D8B144 && dState[1] == 0x60F6C316572983A8 && dState[2] == 0xA2564CA289E5F897 && ((dState[3] ^= 0x00000000CA30DB85) & (0x00000000FFFFFFFF)) == 0) {
				printf("Find Preimage!!! val is %lu.\n",val);
				result[0] = val;
				result[1] = val_sol[1];
				result[2] = val_sol[2];
				result[3] = val_sol[3];

				for (int i = 0; i < 25; i++) {
					printf("%llx ", state_copy[i]);

				}
				printf("\n");
			}

		}
	}

}






int main(int argc, char** argv) {
	tKeccakLane state[25];
	stateInit(state);

	const int para_num = PARA_NUM;
	const int enum_num = ENUM_NUM;


	value_t set_val = atol(argv[1])<<THREADS_SHIFT;

	const int unknown_num = UNKNOWN_NUM;
	const int poly_num = POLY_NUM;

	linearpart_t linear_mat[para_num][poly_num][3];
	linearpart_t working_mat[poly_num][3]; // initialized as the const part of linear matrix. also used as the results of linear part.
	linearpart_t working_mat_copy[poly_num][3];
	linearpart_t working_mat_file[poly_num][3];

	squarepart_t square_mat[para_num][poly_num];
	squarepart_t const_mat[poly_num]; // used to compute the const part from square polys.
	oripoly_t var_all[640][4];


	hipSetDevice(atoi(argv[2]));

	// read the matrix files
	FILE *in1 = fopen("./59/59linear.txt", "r+");
	FILE *in2 = fopen("./59/59square.txt", "r+");
	FILE *in4 = fopen("./59/59working.txt", "r+");
	FILE *in5 = fopen("./59/59totalLinear640.txt", "r+");

	char c1, c2, c4, c5;
	for (int i = 0; i < para_num; i++) {
		for (int j = 0; j < poly_num; j++) {
			linear_mat[i][j][0] = 0;
			linear_mat[i][j][1] = 0;
			linear_mat[i][j][2] = 0;
			square_mat[i][j] = 0;

			for (int k = 0; k < 192; k++) {
				fscanf(in1, "%c", &c1);
				while (c1 != '0' && c1 != '1') {
					fscanf(in1, "%c", &c1);
				}
				if (c1 == '1') {

					linear_mat[i][j][k / 64] ^= ((linearpart_t) 1 << (k % 64));
				}
			}

			for (int k = 0; k < 64; k++) {
				fscanf(in2, "%c", &c2);
				while (c2 != '0' && c2 != '1') {
					fscanf(in2, "%c", &c2);
				}
				if (c2 == '1') {
					square_mat[i][j] ^=
							((squarepart_t) 1 << (para_num - 1 - k));
				}
			}
		}

	}

	for (int i = 0; i < poly_num; i++) {
		working_mat[i][0] = 0;
		working_mat[i][1] = 0;
		working_mat[i][2] = 0;
		for (int j = 0; j < 192; j++) {
			fscanf(in4, "%c", &c4);
			while (c4 != '0' && c4 != '1') {
				fscanf(in4, "%c", &c4);
			}
			if (c4 == '1') {

				working_mat[i][j / 64] ^= ((linearpart_t) 1 << (j % 64));
			}
		}
		working_mat_file[i][0] = working_mat[i][0];
		working_mat_file[i][1] = working_mat[i][1];
		working_mat_file[i][2] = working_mat[i][2];
	}

	for (int i = 0; i < 640; i++) {
		var_all[i][0] = 0;
		var_all[i][1] = 0;
		var_all[i][2] = 0;
		var_all[i][3] = 0;
		for (int j = 0; j < 256; j++) {

			fscanf(in5, "%c", &c5);
			while (c5 != '0' && c5 != '1') {
				fscanf(in5, "%c", &c5);

			}
			if (c5 == '1') {
				var_all[i][j / 64] ^= ((value_t) 1 << (j % 64));
			}

		}

	}

	fclose(in1);
	fclose(in2);
	fclose(in4);
	fclose(in5);

	printf("finish reading file!\n");


	//allocate device memory

	linearpart_t linear_mat_enum[ENUM_NUM * POLY_NUM * 3];
	squarepart_t square_mat_enum[ENUM_NUM * POLY_NUM];
	value_t var_all_enum[640 * 4];

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			for (int k = 0; k < 3; k++) {
				linear_mat_enum[i * POLY_NUM * 3 + j * 3 + k] =
						linear_mat[i][j][k];

			}
		}
	}

	for (int i = 0; i < ENUM_NUM; i++) {
		for (int j = 0; j < POLY_NUM; j++) {
			square_mat_enum[i * POLY_NUM + j] = square_mat[i][j];
		}
	}

	hipMemcpyToSymbol(HIP_SYMBOL(d_linear_mat), linear_mat_enum,
			3 * ENUM_NUM * POLY_NUM * sizeof(linearpart_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_square_mat), square_mat_enum,
			ENUM_NUM * POLY_NUM * sizeof(squarepart_t));

	for (int i = 0; i < 640; i++) {
		for (int j = 0; j < 4; j++) {
			var_all_enum[i * 4 + j] = var_all[i][j];
		}

	}
	hipMemcpyToSymbol(HIP_SYMBOL(d_var_all), var_all_enum, 640 * 4 * sizeof(value_t));

	printf("finish copying device memory!\n");

	hipError_t err = hipSuccess;
	int thidx = BLOCK_NUM * THREAD_NUM;

	value_t *d_val = NULL;
	err = hipMalloc((void **) &d_val, thidx * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t *d_sol_total = NULL;
	err = hipMalloc((void **) &d_sol_total, thidx * 3 * sizeof(value_t));
	if (err != hipSuccess) {
		printf("Failed to allocate device value (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	linearpart_t *d_working_mat_copy = NULL;
	err = hipMalloc((void **) &d_working_mat_copy,
			thidx * poly_num * 3 * sizeof(linearpart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate device working_mat_copy (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	squarepart_t *d_const_mat = NULL;
	err = hipMalloc((void **) &d_const_mat,
			thidx * poly_num * sizeof(squarepart_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices const_mat (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy oripolys from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	value_t h_result[4] = {0, 0, 0, 0};
	value_t *d_result = NULL;
	err = hipMalloc((void **) &d_result, 4 *sizeof(value_t));
	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to allocate devices result (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_result, h_result,
	4 * sizeof(value_t),hipMemcpyHostToDevice);

		if (err != hipSuccess) {
			fprintf(stderr,
					"Failed to copy result from host to device (error code %s)!\n",
					hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}


	printf("finish allocate device memory!\n");


	//deal with the case with all parameters are 0's on CPU
	int s_total_p0 = 0;
	value_t *val_arr = (value_t*)calloc(thidx, sizeof(value_t));
	linearpart_t *working_mat_copy_arr = (linearpart_t*)calloc(thidx * POLY_NUM * 3, sizeof(linearpart_t));
	squarepart_t *const_mat_arr = (squarepart_t*)calloc(thidx * POLY_NUM, sizeof(squarepart_t));
	for (int thi = 0; thi < thidx; thi++) {

		value_t sol_num = 0;
		value_t solutions[SOL_MAX_NUM][3];

		//int sol_total = 0;
		value_t val = (set_val + (value_t) thi) << ENUM_NUM;

		val_arr[thi] = val;
		for (int pi = 0; pi < POLY_NUM; pi++) {
			working_mat[pi][0] = working_mat_file[pi][0];
			working_mat[pi][1] = working_mat_file[pi][1];
			working_mat[pi][2] = working_mat_file[pi][2];

			const_mat[pi] = 0;
		}

		for (int pos = enum_num; pos < para_num; pos++) {

			if (val & ((value_t) 1 << pos)) {

				for (int pi = 0; pi < poly_num; pi++) {
					working_mat[pi][0] ^= linear_mat[pos][pi][0];
					working_mat[pi][1] ^= linear_mat[pos][pi][1];
					working_mat[pi][2] ^= linear_mat[pos][pi][2];
				}

				for (int pi = 0; pi < poly_num; pi++) {
					const_mat[pi] ^= square_mat[pos][pi];

				}

			}

		}


		for (int i = 0; i < poly_num; i++) {
			working_mat_copy[i][0] = working_mat[i][0];
			working_mat_copy[i][1] = working_mat[i][1];
			working_mat_copy[i][2] = working_mat[i][2];

			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i] = working_mat_copy[i][0];
			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i + 1] = working_mat_copy[i][1];
			working_mat_copy_arr[thi * POLY_NUM * 3 + 3 * i + 2] = working_mat_copy[i][2];

			const_mat_arr[thi * POLY_NUM + i] = const_mat[i];

		}

		for (int pi = 0; pi < poly_num; pi++) {

			value_t w = const_mat[pi] & val;

			w = (w) ^ (w >> 32);
			w = (w) ^ (w >> 16);
			w = (w) ^ (w >> 8);
			w = (w) ^ (w >> 4);
			w = (w) ^ (w >> 2);
			w = (w) ^ (w >> 1);

			if (w & (value_t) 1) {

				working_mat[pi][0] ^= (linearpart_t) 1;
			}

		}

		sol_num = gauss_host(working_mat, POLY_NUM, UNKNOWN_NUM, solutions);
		s_total_p0 += sol_num;
		 for (int s = 0; s < sol_num; s++) {

			tKeccakLane state[25];
			stateInit(state);
			getStates(state, var_all,val, solutions[s]);

			if(checkHashValue(state)){

				FILE *out = fopen("preimage_b2_59.txt","a+");
				printf("we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",val,solutions[s][0],solutions[s][1],solutions[s][2]);
				fprintf(out,"we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",val,solutions[s][0],solutions[s][1],solutions[s][2]);
				fclose(out);
			}

		    }


	}

	printf("finish cpu computing!\n");

	//begin device part: copy value from host ot devide
	err = hipMemcpy(d_val, val_arr, thidx * sizeof(value_t),
			hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("Failed to copy value from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_working_mat_copy, working_mat_copy_arr,
			thidx * 3 * poly_num * sizeof(linearpart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy working_mat_copy from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_const_mat, const_mat_arr,
			thidx * poly_num * sizeof(squarepart_t), hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy const_mat from host to device (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("enum num : %d\nblock num : %d\nthread num : %d\n", ENUM_NUM,
			BLOCK_NUM, THREAD_NUM);


	//solve linear system on GPU
	hipEvent_t start1;
	hipEventCreate(&start1);
	hipEvent_t stop1;
	hipEventCreate(&stop1);
	hipEventRecord(start1, NULL);

	printf("begin solve linear system!\n");
	solveLinear<<<BLOCK_NUM, THREAD_NUM>>>(d_working_mat_copy, d_const_mat,
			d_val, d_sol_total,d_result);

	hipEventRecord(stop1, NULL);
	hipEventSynchronize(stop1);
	float msecTotal1 = 0.0f;
	hipEventElapsedTime(&msecTotal1, start1, stop1);
	err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch solveLinear kernel (error code %s)!\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	value_t h_sol_total[thidx];
	err = hipMemcpy(h_sol_total, d_sol_total, thidx * sizeof(value_t),
				hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
				"Failed to copy total solution numbers from device to host (error code %s)!\n",
				hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}

	err = hipMemcpy(h_result, d_result,4 * sizeof(value_t),
					hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		fprintf(stderr,
					"Failed to copy result from device to host (error code %s)!\n",
					hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//print the results

	if(h_result[0]!=0 || h_result[1]!=0 || h_result[2]!=0 ||  h_result[3]!=0){
	    	FILE *out = fopen("preimage_b2_59.txt","a+");
			printf("we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",h_result[0],h_result[1],h_result[2],h_result[3]);
			fprintf(out,"we have done on GPU!!! val:%lu, sol:%lu %lu %lu\n",h_result[0],h_result[1],h_result[2],h_result[3]);
			fclose(out);
		}else{
			FILE *out = fopen("result.txt","a+");
			long sol_all_threads = s_total_p0;
			for(int i = 0;i < thidx;i++){
				sol_all_threads += h_sol_total[i];
			}
			printf("val : %lu~%lu ,find %lu solutions, none is correct...\n",set_val << ENUM_NUM ,(set_val << ENUM_NUM)+(THREAD_NUM * BLOCK_NUM) * (1 << ENUM_NUM) -1, sol_all_threads);
			fprintf(out, "Part %d finished -- val : %lu~%lu ,find %lu solutions, none is correct...\n",atol(argv[1]), set_val << ENUM_NUM ,(set_val << ENUM_NUM)+(THREAD_NUM * BLOCK_NUM) * (1 << ENUM_NUM) -1, sol_all_threads);
			fclose(out);

		}


	printf("time:%.3lf ms\n---------------------------------------\n", msecTotal1);


	hipFree(val_arr);
	hipFree(d_working_mat_copy);
	hipFree(d_const_mat);
	hipFree(d_val);
	hipFree(d_sol_total);
	hipFree(d_result);
}
